#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

__global__ void reduce0(int * input, int *output, int N)
{
	extern __shared__ int sdata[1024];

	////printf("%d, %d\n", tid, N);

	//int *idata = input + blockIdx.x*blockDim.x;
	
	//set thread ID
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockDim.x*blockIdx.x * 2 + threadIdx.x;
	//boundary check
	if (tid >= N) return;

	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	
	//if (idx + blockDim.x < N)
	//{
	//	input[idx] += input[idx + blockDim.x];
	//}

	//convert global data pointer to the
	int *idata = input + blockIdx.x*blockDim.x * 2;
	if (idx + blockDim.x<N)
	{
		input[idx] += input[idx + blockDim.x];
		//printf("idx is: %d, blockDim is: %d, N is: %d\n", idx, blockDim.x, N);
	}
	__syncthreads();

	//sdata[tid] = input[i];
	__syncthreads();

	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			idata[tid] += idata[tid + stride];
		}
		//synchronize within block
		__syncthreads();
	}

	if (tid == 0)
	{
		output[blockIdx.x] = idata[0];
	}
}

int gpu_reduce0(int * input, int *output, int N, dim3 grid, dim3 block)
{
	int * cpu_sum = new int[1];
	int * gpu_sum;
	hipMalloc(&gpu_sum, sizeof(int));
	reduce0<<<grid.x/2, block >>>(input, output, N);
	hipDeviceSynchronize();
	int * cpu_output = new int[grid.x];
	hipMemcpy(cpu_output, output, grid.x * sizeof(int), hipMemcpyDeviceToHost);

	int s = 0;
	for (int i = 0; i < grid.x; i++)
	{
		s += cpu_output[i];
	}

	/*reduce0 << <1, grid.x >> > (output, gpu_sum, grid.x);
	hipDeviceSynchronize();
	hipMemcpy(cpu_sum, gpu_sum, sizeof(int), hipMemcpyDeviceToHost);
	return cpu_sum[0];*/
	return s;
}